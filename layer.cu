#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <random>

#include "hipblas.h"
#include <hip/hip_runtime.h>


#define IDX2C(i, j, ld) ((( j )*( ld ))+( i )) // ld - leading dimension

std::default_random_engine generator;
std::normal_distribution<float> distribution(0.0, 0.5);

void initialization(float* a, int size) {
	for (int i = 0; i < size; i++) {
		a[i] = distribution(generator);
		//a[i] = i;
	}
}

__global__ void reluHelper(float* Z, float* dZ, int numElements) {
	// perform relu activation and calculate gradients simultaneously
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		if (Z[i] < 0) {
			Z[i] = 0;
			dZ[i] = 0;
		}
		else {
			dZ[i] = 1;
		}
	}
}

__global__ void elementMulHelper(float* A, float* B, int numElements, bool invB) {
	// perform relu activation and calculate gradients simultaneously
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		if (invB) {
			A[i] /= B[i];
		}
		else {
			A[i] *= B[i];
		}
	}
}

__global__ void expHelper(float* A, int numElements) {
	// perform element-wise exp
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		A[i] = exp(A[i]);
	}
}

__global__ void elementAddHelper(float* A, float* B, float alpha, int numElements) {
	// perform element-wise addition
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		A[i] += alpha * B[i];
	}
}

void printMatrix(float* a, int r, int c) {
	// print matrix row by row, debugging purpose
	// r - number of rows
	// c - number of columns
	for (int i = 0; i < r; i++) {
		for (int j = 0; j < c; j++) {
			printf(" %6.3f", a[IDX2C(i, j, r)]);
		}
		printf("\n");
	}
}

class Layer {
	protected:
		int l_prev, l_curr; // l_prev : neural number of previous layer, l : neural number of current layer
		float* W;  // weights, matrix of size (l_curr, l_prev)
		float* dW; // W gradients
		float* b;  // bias, vector of size (l_curr,)
		float* db; // b gradients
		float* dZ; // gradient of activation
		           // in forward pass, store dA/dZ
		           // in backward pass, dL/dZ = dL/dA * dA/dZ
		const float* A_prev; // activation from previous layer, book-keeping for backward pass

	public:
		Layer(int l1, int l2) {
			l_prev = l1;
			l_curr = l2;

			// allocate memory
			W  = (float *)malloc(l1 * l2 * sizeof(float));
			dW = (float *)malloc(l1 * l2 * sizeof(float));
			b  = (float *)malloc(l2 * sizeof(float));
			db = (float *)malloc(l2 * sizeof(float));

			// initialize W and b
			initialization(W, l1 * l2);
			initialization(b, l2);
			printf("W matrix:\n");
			printMatrix(W, l_curr, l_prev);
			printf("b:\n");
			printMatrix(b, l_curr, 1);
		}

		float* forward(const float* X_in, int batch) {
			// X_in input matrix, size (l_prev, batch_size), each column is a data point
			A_prev = X_in; // save activation from previous layer for backprop
			float* X_out = WX_b(W, X_in, b, l_curr, batch, l_prev); // X_out = Z = W @ X + b
			printf("Z:\n");
			printMatrix(X_out, l_curr, batch);

			// allocate memory for dZ (gradient of activation) and perform activation
			int numElements = l_curr * batch;
			dZ = (float *)malloc(numElements * sizeof(float));
			relu(numElements, X_out, dZ);
			printf("dZ:\n");
			printMatrix(dZ, l_curr, batch);

			return X_out; // X_out = A = relu(Z)
		}

		float* backward(const float* dA, int batch) {
			// dA input matrix, size (l_curr, batch_size), each column is gradient of a datapoint of current layer
			// dA_prev output matrix
			float* dA_prev;
			int numElements = l_prev * batch;
			dA_prev = (float *)malloc(numElements * sizeof(float));

			// calculate dZ, dW, db, dA_prev
			elementwiseMul(l_curr * batch, dZ, dA, false);                                        // dZ = dL/dA * dA/dZ
			printf("dZ:\n");
			printMatrix(dZ, l_curr, batch);

			matrixMul(dZ, A_prev, dW, l_curr, l_prev, batch, false, true, 1/(float)batch, 0.f); // dW = dL/dZ * dZ/dW = 1/m * dZ @ A_prev.T
			printf("dW:\n");
			printMatrix(dW, l_curr, l_prev);

			reduceSum(dZ, db, l_curr, batch, false);                                              // db = dL/dZ * dZ/db = 1/m * sum(dZ, axis=1)
			printf("db:\n");
			printMatrix(db, l_curr, 1);

			matrixMul(W, dZ, dA_prev, l_prev, batch, l_curr, true, false, 1.0f, 0.f);      // dA_prev = dL/dZ * dZ/dA_prev = W.T @ dZ

			return dA_prev;
		}

		void gradientUpdate(float alpha) {
			// perform parameter update w.r.t to gradient direction with learning rate alpha
			elementwiseAdd(l_curr * l_prev, W, dW, -alpha);
			elementwiseAdd(l_curr, b, db, -alpha);

			printf("W matrix:\n");
			printMatrix(W, l_curr, l_prev);
			printf("b:\n");
			printMatrix(b, l_curr, 1);
		}

		void freeMemory() {
			// release memory
			free(W);
			free(dW);
			free(b);
			free(db);
			free(dZ);
			free((float*)A_prev);
		}

	protected:
		// helper functions
		float* WX_b(const float* W, const float* X, const float* b, int m, int n, int k) {
			// perform W @ X + b in a batch
			// m - l_curr, n - batch_size, k - l_prev
			// W is matrix of size (m, k)
			// X is matrix of size (k, n)
			// b is vector of size (m,)
			// c is matrix of size (m, n)
			float * c;
			c = (float *)malloc(m * n * sizeof(float)); // allocate memory for c
			broadcast(c, b, m, n, true); // broadcast b

			matrixMul(W, X, c, m, n, k, false, false, 1.0f, 1.0f);

			return c;
		}

		void matrixMul(const float* A, const float* B, float* C, int m, int n, int k, bool transA, bool transB, float alpha, float beta) {
			// C = op(A) @ op(B) + beta * C
			// op(A) is matrix of size (m, k)
			// op(B) is matrix of size (k, n)
			//   C   is matrix of size (m, n)
			// modifies content of C in-place

			hipblasHandle_t handle; // CUBLAS context

			float * d_a; // d_a - a on the device
			float * d_b; // d_b - b on the device
			float * d_c; // d_c - c on the device

			hipMalloc((void **)& d_a, m*k * sizeof(*A)); // device
			hipMalloc((void **)& d_b, k*n * sizeof(*B)); // device
			hipMalloc((void **)& d_c, m*n * sizeof(*C)); // device

			hipblasCreate(&handle); // initialize CUBLAS context

			// copy matrices from the host to the device
			hipblasSetMatrix(m, k, sizeof(*A), A, m, d_a, m); //a -> d_a
			hipblasSetMatrix(k, n, sizeof(*B), B, k, d_b, k); //b -> d_b
			hipblasSetMatrix(m, n, sizeof(*C), C, m, d_c, m); //c -> d_c

			// matrix - matrix multiplication : d_c = alpha * op(d_a) @ op(d_b) + beta * d_c
			// op(d_a) - m x k matrix , op(d_b) - k x n matrix , d_c - m x n matrix
			// alpha, beta read from argument
			hipblasOperation_t opA = HIPBLAS_OP_N;
			hipblasOperation_t opB = HIPBLAS_OP_N;
			int lda = m;
			int ldb = k;
			if (transA) {
				opA = HIPBLAS_OP_T;
				lda = k;
			}
			if (transB) {
				opB = HIPBLAS_OP_T;
				ldb = n;
			}
			hipblasSgemm(handle, opA, opB, m, n, k, &alpha, d_a, lda, d_b, ldb, &beta, d_c, m);

			// copy matrix from device to host
			hipblasGetMatrix(m, n, sizeof(*C), d_c, m, C, m); // cp d_c - >c

			hipFree(d_a); // free device memory
			hipFree(d_b); // free device memory
			hipFree(d_c); // free device memory
			hipblasDestroy(handle); // destroy CUBLAS context
		}

		void reduceSum(float* A, float* b, int l, int batch, bool columnwise) {
			// reduce sum row-wise or column-wise
			// store results in b
			// get b by matrix - vector multiplication
			
			// create a vector of same size as b filled with 1
			float* x;
			float alpha;
			if (columnwise) {
				x = (float *)malloc(l * sizeof(float));
				alpha = 1.0f;
				for (int i = 0; i < l; i++)
					x[i] = 1;

				matrixMul(A, x, b, batch, 1, l, true, false, alpha, 0.f);
			}
			else {
				x = (float *)malloc(batch * sizeof(float));
				alpha = 1 / (float)batch;
				for (int i = 0; i < batch; i++)
					x[i] = 1;

				matrixMul(A, x, b, l, 1, batch, false, false, alpha, 0.f);
			}

			free(x);
		}

		void elementwiseMul(int numElements, float* A, const float* B, bool invB) {
			// element-wise matrix multiplication, store results in A
			// A = A * B
			unsigned int mem_size = numElements * sizeof(*A);

			float * d_A; // d_A - A on the device
			float * d_B; // d_B - B on the device

			hipMalloc((void **)& d_A, mem_size); // device
			hipMalloc((void **)& d_B, mem_size); // device

			// copy matrices from host to device
			hipMemcpy(d_A, A, mem_size, hipMemcpyHostToDevice); // A -> d_A
			hipMemcpy(d_B, B, mem_size, hipMemcpyHostToDevice); // B -> d_B

			int threadsPerBlock = 256;
			int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
			elementMulHelper<<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, numElements, invB);

			// copy A from device to host
			hipMemcpy(A, d_A, mem_size, hipMemcpyDeviceToHost); // d_A -> A

			hipFree(d_A); // free device memory
			hipFree(d_B); // free device memory
		}

		void elementwiseAdd(int numElements, float* A, float* B, float alpha) {
			// element-wise matrix/vector addtion
			// A = A + alpha * B
			unsigned int mem_size = numElements * sizeof(*A);

			float * d_A; // d_A - A on the device
			float * d_B; // d_B - B on the device

			hipMalloc((void **)& d_A, mem_size); // device
			hipMalloc((void **)& d_B, mem_size); // device

			// copy matrices from host to device
			hipMemcpy(d_A, A, mem_size, hipMemcpyHostToDevice); // A -> d_A
			hipMemcpy(d_B, B, mem_size, hipMemcpyHostToDevice); // B -> d_B

			int threadsPerBlock = 256;
			int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
			elementAddHelper<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, alpha, numElements);

			// copy A from device to host
			hipMemcpy(A, d_A, mem_size, hipMemcpyDeviceToHost); // d_A -> A

			hipFree(d_A); // free device memory
			hipFree(d_B); // free device memory
		}

		void relu(int numElements, float* Z, float* dZ) {
			// perform relu activation and calculate gradients simultaneously
			unsigned int mem_size = numElements * sizeof(*Z);

			float * d_Z;  // d_Z  - Z on the device
			float * d_dZ; // d_dZ - dZ on the device

			hipMalloc((void **)& d_Z, mem_size);  // device
			hipMalloc((void **)& d_dZ, mem_size); // device

			// copy Z from host to device
			hipMemcpy(d_Z, Z, mem_size, hipMemcpyHostToDevice); // Z -> d_Z

			int threadsPerBlock = 256;
			int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
			reluHelper<<<blocksPerGrid, threadsPerBlock>>>(d_Z, d_dZ, numElements);

			// copy Z from device to host
			hipMemcpy(Z, d_Z, mem_size, hipMemcpyDeviceToHost);   // d_Z  -> Z (A)
			hipMemcpy(dZ, d_dZ, mem_size, hipMemcpyDeviceToHost); // d_dZ -> dZ

			hipFree(d_Z);  // free device memory
			hipFree(d_dZ); // free device memory
		}

		void broadcast(float* c, const float* b, int l, int batch, bool row) {
			// broadcast bias in a batch
			// c - output matrix of size (l, batch_size)
			// b - vector of size (l, 1) if row
			//     vector of size (batch, 1) if column
			if (row) {
				for (int i = 0; i < l; i++) {
					for (int j = 0; j < batch; j++) {
						c[IDX2C(i, j, l)] = b[i];
					}
				}
			}
			else {
				for (int i = 0; i < l; i++) {
					for (int j = 0; j < batch; j++) {
						c[IDX2C(i, j, l)] = b[j];
					}
				}
			}
			printf("initial c matrix:\n");
			printMatrix(c, l, batch);
		}
};

class SoftmaxLayer: public Layer{
	float* P;

    public:
		SoftmaxLayer(int l1, int l2) : Layer(l1, l2) {}

		float* forward(const float* X_in, int batch) {
			// X_in input matrix, size (l_prev, batch_size), each column is a data point
			A_prev = X_in; // save activation from previous layer for backprop
			float* X_out = WX_b(W, X_in, b, l_curr, batch, l_prev); // X_out = Z = W @ X + b
			printf("Z:\n");
			printMatrix(X_out, l_curr, batch);

			// allocate memory for P (prob matrix) and perform activation
			int numElements = l_curr * batch;
			P = (float *)malloc(numElements * sizeof(float));
			softmax(numElements, X_out);
			P = X_out; // store P for backprop

			return X_out; // X_out = softmax(Z)
		}

	private:
		void softmax(int numElements, float* Z) {
			// softmax operation over each coloum of Z
			// store gradients in dZ
			// 1st x = exp(x) for each element x in Z
			// 2nd p = sum(x) for each column in Z
			// 3rd x = x/p for each element each column in Z
			float* d_Z;
			int batch = numElements / l_curr;
			unsigned int mem_size = numElements * sizeof(*Z);

			// allocate memory for Z on device
			hipMalloc((void **)& d_Z, mem_size);
			hipMemcpy(d_Z, Z, mem_size, hipMemcpyHostToDevice); // Z -> d_Z

			// 1st x = exp(x) for each element x in Z
			int threadsPerBlock = 256;
			int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
			expHelper<<<blocksPerGrid, threadsPerBlock>>>(d_Z, numElements);

			hipMemcpy(Z, d_Z, mem_size, hipMemcpyDeviceToHost);   // d_Z  -> Z
			hipFree(d_Z);  // free device memory
			printf("exp:\n");
			printMatrix(Z, l_curr, batch);

			// 2nd p = sum(x) for each column in Z
			float* p; // vector length batch_size
			p = (float *)malloc(batch * sizeof(float));
			reduceSum(Z, p, l_curr, batch, true);
			printf("p:\n");
			printMatrix(p, batch, 1);

			// 3rd x = x/p for each element each column in Z
			float * P; // probability matrix of size (l_curr, batch)
			P = (float *)malloc(numElements * sizeof(float));
			broadcast(P, p, l_curr, batch, false);
			elementwiseMul(numElements, Z, P, true);
			free(p);
			free(P);
		}
};

int main() {
	int batch = 5;
	int feature = 4;
	int l1 = 3;
	int l2 = 2;

	float* X;
	X = (float *)malloc(feature * batch * sizeof(float));
	initialization(X, feature * batch);
	printf("foward pass\n");
	printf("input matrix:\n");
	printMatrix(X, feature, batch);

	Layer l = Layer(feature, l1);
	SoftmaxLayer s = SoftmaxLayer(l1, l2);
	float* X1 = l.forward(X, batch);
	printf("output matrix:\n");
	printMatrix(X1, l1, batch);

	float* X2 = s.forward(X1, batch);
	printf("output matrix:\n");
	printMatrix(X2, l2, batch);
	/*
	printf("\nbackward pass\n");
	float* dA;
	dA = (float *)malloc(l1 * batch * sizeof(float));
	initialization(dA, l1 * batch);
	printf("input matrix:\n");
	printMatrix(dA, l1, batch);

	float* dA0 = l.backward(dA, batch);
	printf("output matrix:\n");
	printMatrix(dA0, feature, batch);

	printf("\ngradient updata\n");
	l.gradientUpdate(1);
	l.freeMemory(); */
}